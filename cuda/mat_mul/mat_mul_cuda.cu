
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void gpuMatMul(float * A, float * B, float *C,
                          int ROW_A, int COL_A, int COL_B) {
  /******************** TODO *********************/
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int k;
    float sum = 0.0f;

    if(i < ROW_A && j < COL_B){
        for(k = 0; k < COL_A; k++){
            sum += A[i * COL_A + k] * B[k * COL_B + j ];
        }
        C[i * COL_B + j] = sum;
    }
}

void mat_mul_cuda(float *A, float *B, float *C,
                    int ROW_A, int COL_A, int COL_B) {
  /******************** TODO *********************/
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, sizeof(float) * ROW_A * COL_A);
    hipMalloc(&d_B, sizeof(float) * COL_A * COL_B);
    hipMalloc(&d_C, sizeof(float) * ROW_A * COL_B);

    hipMemcpy(d_A, A, sizeof(float) * ROW_A * COL_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, sizeof(float) * COL_A * COL_B, hipMemcpyHostToDevice);

    dim3 dimBlock(16, 16);
    dim3 dimGrid(COL_B / 16, ROW_A / 16);
    gpuMatMul<<< dimGrid, dimBlock >>> (d_A, d_B, d_C, ROW_A, COL_A, COL_B);

    hipMemcpy(C, d_C, sizeof(float) * ROW_A * COL_B, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

