
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void gpuVecAdd(float *A, float *B, float *C) {
    // TODO: write kernel code here
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    C[tid] = A[tid] + B[tid];
}

void init(float *V, int N) {
    for (int i = 0; i < N; i++) {
        V[i] = rand() % 100;
    }
}

void verify(float *A, float *B, float *C, int N) {
    for (int i = 0; i < 16384; i++) {
        if (A[i] + B[i] != C[i]) {
            printf("Verification failed! A[%d] = %d, B[%d] = %d, C[%d] = %d\n",
                    i, A[i], i, B[i], i, C[i]);
            return;
        }
    }
    printf("Verification success!\n");
}

int main() {
    int N = 16384;

    float *A = (float*)malloc(sizeof(float) * N);
    float *B = (float*)malloc(sizeof(float) * N);
    float *C = (float*)malloc(sizeof(float) * N);

    init(A, N);
    init(B, N);

    // Memory objects of the device
    float *d_A, *d_B, *d_C;

    // TODO: allocate memory objects d_A, d_B, and d_C.
    hipMalloc(&d_A, sizeof(float) * N);
    hipMalloc(&d_B, sizeof(float) * N);
    hipMalloc(&d_C, sizeof(float) * N);
    // TODO: copy "A" to "d_A" (host to device).
    hipMemcpy(d_A, A, sizeof(float) * N, hipMemcpyHostToDevice);
    // TODO: copy "B" to "d_B" (host to device).
    hipMemcpy(d_B, B, sizeof(float) * N, hipMemcpyHostToDevice);

    // TODO: launch the kernel.
    dim3 dimBlock(32, 1);
    dim3 dimGrid(N/32, 1);
    gpuVecAdd<<< dimGrid, dimBlock >>> (d_A, d_B, d_C);
    // TODO: copy "d_C" to "C" (device to host).
    hipMemcpy(C, d_C, sizeof(float) * N, hipMemcpyDeviceToHost);

    verify(A, B, C, N);

    // TODO: release d_A, d_B, and d_C.
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(A);
    free(B);
    free(C);

    return 0;
}

